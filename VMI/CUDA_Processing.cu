#include "hip/hip_runtime.h"
//////////////////////////////////////////////
//                                          //
//  delocalisation of the data treatement   //
//     on the GPU with CUDA framework       //
//                                          //
//////////////////////////////////////////////

#include "CUDA_Processing.hpp"

/*---------------- GLOBAL VARIABLES TO BE USED ON THE GPU ------------------------------*/




/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------- CUDA KERNELS DEFINITIONS ---------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/

__global__ void InitParam(long *d_FrameParamPtr)
{
    d_FrameParamPtr[6]=0; // number of counts per frame
    __syncthreads();
}

__global__ void ThresholdingData(unsigned char *src,unsigned int *srcAcc, long *d_FrameParamPtr,unsigned int *SSDataStream,unsigned int *SSIndexStream,unsigned int *BlockCountBuff,int t, unsigned char *d_BGCorr)
{
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    unsigned int BlockThreadoffset;
    unsigned int Blockoffset;
    __shared__ unsigned int BCB;
    BCB=BlockCountBuff[t];

    long SingleShotRecord=d_FrameParamPtr[7];
    int thresh=(int) d_FrameParamPtr[2];
    long counting_mode=d_FrameParamPtr[4];
    
    
    //if(t==0 && d_FrameParamPtr[0]==0 && d_FrameParamPtr[5]%d_FrameParamPtr[1]==0){srcAcc[i]=0;}

    //__syncthreads();

    /*--------------------------------------------------------------------------*/
    /*--------------------------- BG SUBSTRACTION ------------------------------*/
    /*--------------------------------------------------------------------------*/
    
    int u =(int) src[i]-(int) d_BGCorr[i];
    if(u<0){src[i]=0;}
    else{src[i]=(unsigned int) u;} 
    
    __syncthreads();

    /*--------------------------------------------------------------------------*/
    /*--------------------------- THRESHOLDING DATA ----------------------------*/
    /*--------------------------------------------------------------------------*/

    if(src[i]<thresh)
    {
        src[i]=0;
        __syncthreads();
     
    }
    else{src[i]=src[i]-(thresh-1);__syncthreads();}
    
    __syncthreads();
    
    /*--------------------------------------------------------------------------------*/
    /*---------------------------END OF THRESHOLDING DATA ----------------------------*/
    /*--------------------------------------------------------------------------------*/
    
    /*-----------------------------------------------------------------------------*/
    /*---------------------------- CENTROIDING THE DATA ---------------------------*/
    /*-----------------------------------------------------------------------------*/
    
    if(counting_mode==1)
    {
        bool MaxPixl=0;
        int NcountFrame=0;
        //////////////////////////////////////////////////
        // block    -3   -2    1    0    1    2    3    //
        // thread                                       //
        //   -3                    im3                  //
        //   -2               hm2  im2  jm1             //
        //   -1          gm1  hm1  im1  jm1  km1        //
        //    0      f    g    h    i    j    k    l    //
        //    1          gp1  hp1  ip1  jp1  kp1        //
        //    2               hp2  ip2  jp2             //
        //    3                    ip3                  //
        //////////////////////////////////////////////////
        
    
        if (threadIdx.x > 2 || blockIdx.x > 2 || threadIdx.x <(blockDim.x-2) || blockIdx.x < (blockDim.x-2) )
        {
            
               int ip1=(threadIdx.x+1)+blockIdx.x*blockDim.x;
               //int ip2=(threadIdx.x+2)+blockIdx.x*blockDim.x;
               //int ip3=(threadIdx.x+3)+blockIdx.x*blockDim.x;
               int im1=(threadIdx.x-1)+blockIdx.x*blockDim.x;
               //int im2=(threadIdx.x-2)+blockIdx.x*blockDim.x;
               //int im3=(threadIdx.x-3)+blockIdx.x*blockDim.x;
             
               int h=threadIdx.x+(blockIdx.x-1)*blockDim.x;
               int hp1=(threadIdx.x+1)+(blockIdx.x-1)*blockDim.x;
               //int hp2=(threadIdx.x+2)+(blockIdx.x-1)*blockDim.x;
               int hm1=(threadIdx.x-1)+(blockIdx.x-1)*blockDim.x;
               //int hm2=(threadIdx.x-2)+(blockIdx.x-1)*blockDim.x;
             
               int j=threadIdx.x+(blockIdx.x+1)*blockDim.x;
               int jp1=(threadIdx.x+1)+(blockIdx.x+1)*blockDim.x;
               //int jp2=(threadIdx.x+1)+(blockIdx.x+1)*blockDim.x;
               int jm1=(threadIdx.x-1)+(blockIdx.x+1)*blockDim.x;
               //int jm2=(threadIdx.x+1)+(blockIdx.x+1)*blockDim.x;
             
               //int k=threadIdx.x+(blockIdx.x+2)*blockDim.x;
               //int kp1=(threadIdx.x+1)+(blockIdx.x+2)*blockDim.x;
               //int km1=(threadIdx.x-1)+(blockIdx.x+2)*blockDim.x;
             
               //int g=threadIdx.x+(blockIdx.x-2)*blockDim.x;
               //int gp1=(threadIdx.x+1)+(blockIdx.x-2)*blockDim.x;
               //int gm1=(threadIdx.x-1)+(blockIdx.x-2)*blockDim.x;
              
               //int l=threadIdx.x+(blockIdx.x+3)*blockDim.x;
              
               //int f=threadIdx.x+(blockIdx.x-3)*blockDim.x;
              
             /* char C=src[i];*/
                if(src[i]>src[ip1] && src[i]>src[im1] && src[i]>src[j] && src[i]>src[jp1] && src[i]>src[jm1] && src[i]>src[h] && src[i]>src[hp1] && src[i]>src[hm1])
                {
                	MaxPixl=true;
                   // NcountFrame+=1;
                    d_FrameParamPtr[6]+=1;
                }
                else
                {
                    MaxPixl=false;
                }
            
        }
        else
        {
            MaxPixl=false;
        }
        __syncthreads();

        //if(i==1){d_FrameParamPtr[6]=NcountFrame;} // only one thread writes down the number of counts

        if(MaxPixl==true)
        {
            src[i]=1;
        }
        else
        {
            src[i]=0;
        }
         __syncthreads();
    }
    
    /*------------------------------------------------------------------------------------*/
    /*---------------------------- END OF CENTROIDING THE DATA ---------------------------*/
    /*------------------------------------------------------------------------------------*/

    /*------------------------------------------------------------------------------------*/
    /*----------------------------- ACCUMULATE DATA ON FRAME -----------------------------*/
    /*------------------------------------------------------------------------------------*/

    //if(t==0){

        srcAcc[i]=srcAcc[i]+(unsigned int) src[i];
        __syncthreads();
    
    //}
   
    /*------------------------------------------------------------------------------------*/
    /*-------------------------- END OF ACCUMULATE DATA ON FRAME -------------------------*/
    /*------------------------------------------------------------------------------------*/
    
    
    /*------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
    /*------------------------- VECTOR COMPACTION IN THE KERNEL (ref D.M. Hughes & al. Computer Graphics forum, Vol 32, iss 6, p178-188 (2013)) ------------------------*/
    /*------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
    if(SingleShotRecord==1)
    {
    __shared__ unsigned int W[25];          // array of warp offsets (for 800 threads/block -> 25 warps)
    
    
    unsigned int warpindex=i/32;                    // will take only the floor of the value due to the fact that it is defined as int.
    unsigned int threadindexinwarp= i%32;
    unsigned int threadmask=INT_MAX>>(32-threadindexinwarp); // for example: [1111111111]->[0011111111] if arg=2;
    
    unsigned int ballotresult=__ballot(src[i]) & threadmask;
    unsigned int threadoffset=__popc(ballotresult);
    
    if(threadindexinwarp==31)
    {
        if(src[i]==0)
        {
            W[warpindex]=threadoffset;
        }
        else
        {
            W[warpindex]=threadoffset+1;
        }
        __threadfence();
        __syncthreads();
    }
    
    BlockThreadoffset=__syncthreads_count(src[i]);
    
    unsigned int warpoffset=(__popc(__ballot(W[warpindex] & 1)&threadmask)>>warpindex) +(__popc(__ballot(W[warpindex]&2)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&4)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&8)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&16)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&32)&threadmask)>>warpindex);
    
    Blockoffset=atomicAdd(&BCB,BlockThreadoffset);        // Last problem on that line in compilation.
    BlockCountBuff[t]=BCB;
    unsigned int Totaloffset;
    
    Totaloffset=Blockoffset+warpoffset+threadoffset;
    
    if(src[i]>0)
    {
        SSDataStream[Totaloffset]=src[i];
        SSIndexStream[Totaloffset]=i;
    }
    
    __syncthreads();
    }
    /*-----------------------------------------------------------------------------*/
    /*------------------------- END OF VECTOR COMPACTION --------------------------*/
    /*-----------------------------------------------------------------------------*/
      
}

__global__ void ResetDataArrays(unsigned int *srcAcc)
{
     int i=threadIdx.x+blockIdx.x*blockDim.x;
     srcAcc[i]=0;
     __syncthreads();
}


/*\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-*/


/*----------------------------------------------------------------------------------------*/
/*---------------------------- END OF CUDA KERNELS DEFINITIONS ---------------------------*/
/*----------------------------------------------------------------------------------------*/
/*------------------------------------ DEFINITION OF -------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*-------------------------------- C++ WRAPPING FUNCTIONS --------------------------------*/
/*----------------------------------------------------------------------------------------*/


hipError_t InitialiseCUDAMem(unsigned char **& h_StreamPtr, unsigned int **& d_SSDataStream_ptr, unsigned int **& d_SSIndexStream_ptr, long *& d_FrameParamPtr, unsigned char *& d_FramePtr, unsigned int *& d_PicturePtr, int Nbytes, unsigned char *& d_BGCorr)
{
    hipError_t cudaStatus;
    
    /* Pinned Host memory allocation */

    for(int i=0;i<5;i++)
    {
    cudaStatus=hipHostMalloc((void**)&h_StreamPtr[i],Nbytes);
    }

    //hipHostMalloc((void**)&h_StreamAccPtr,Nbytes*4);
    
    /* Single shot data array allocation */
    /*(for a maximum of 50 millions points which represents 5millions frames at 10 evt/frame (2h45min acq)) */
    
    for(int i=0;i<5;i++)
    {
    hipMalloc((void**) &d_SSDataStream_ptr[i],40000000);
    hipMalloc((void**) &d_SSIndexStream_ptr[i],40000000);
    }
    
    /* memory allocation for frame on GPU */
    
    hipMalloc((void**) &d_FrameParamPtr,11*8);
    hipMalloc((void**) &d_FramePtr,Nbytes);
    hipMalloc((void**) &d_PicturePtr,Nbytes*4);

    hipMalloc((void**) &d_BGCorr,Nbytes);
    
    return cudaStatus;
}

hipError_t FreeCUDAMem(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr, long *d_FrameParamPtr, unsigned char *d_FramePtr, unsigned int *d_PicturePtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    for(int i=0;i<5;i++)
    {
        hipHostFree(h_StreamPtr[i]);
        hipFree(d_SSDataStream_ptr[i]);
        hipFree(d_SSIndexStream_ptr[i]);
    }

    cudaStatus=hipFree(d_FrameParamPtr);
    hipFree(d_FramePtr); // Free the GPU ressources.
    hipFree(d_PicturePtr);
    hipFree(d_BGCorr);
    
    return cudaStatus;
}

hipError_t CUDAresetDataArrays(unsigned int *d_PicturePtr)
{
    hipError_t cudaStatus;
    dim3 grid(400,1);
    dim3 threads(400,1);

    ResetDataArrays<<<grid,threads>>>(d_PicturePtr);
    cudaStatus=hipDeviceSynchronize();
    
    return cudaStatus;
        
}

hipError_t CUDAbackgroundFrameToGPU(unsigned char *h_BGCorr, unsigned char *d_BGCorr, int Nbytes)
{   
    hipError_t cudaStatus;
    cudaStatus=hipMemcpy(d_BGCorr,h_BGCorr,Nbytes,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return cudaStatus;
}

hipError_t CUDAProcessingData(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr,unsigned int *SharedMem, unsigned char *d_FramePtr,  unsigned int *d_PicturePtr, int Nbytes, long *h_FrameParamPtr, long *d_FrameParamPtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    /*---- Cuda streams to optimize data transfer and kernel treatement (5 streams for now) ----*/
    
    hipStream_t *stream=new hipStream_t[5];
    
    for(int i=0;i<5;i++)
    {
        hipStreamCreate(&stream[i]);
    }

    /*---- Buffers for frame Compaction ----*/
    
    unsigned int *d_BlockCountBuff;
    hipMalloc((void**) &d_BlockCountBuff,5*4);
    //__device__ unsigned int d_BlockCountBuff[5];
    
    
    
    /*---- Parameters copy from GUI ----*/
    
    hipMemcpy(d_FrameParamPtr,h_FrameParamPtr,11*8,hipMemcpyHostToDevice);  // copy parameters to treat the data
    
    dim3 grid(160,1);
    dim3 threads(1000,1);
    
    unsigned int* d_FrameIndexPtr;
    hipMalloc((void**) &d_FrameIndexPtr,Nbytes*4);

    hipEvent_t* event=new hipEvent_t[10];

    for(int i=0;i<10;i++)
    {
    hipEventCreate(&event[i]);
	}

    /*---- Core of the processing: data transfer then kernel execution ----*/

    if(h_FrameParamPtr[1]<5)
    {
        for(int t=0;t<h_FrameParamPtr[1];t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);

                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                //hipDeviceSynchronize();
                hipEventRecord(event[h_FrameParamPtr[1]],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                //hipDeviceSynchronize();
                hipEventRecord(event[t+h_FrameParamPtr[1]],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    else
    {
        for(int t=0;t<5;t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);
        
                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                //hipDeviceSynchronize();
                hipEventRecord(event[5],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                //hipDeviceSynchronize();
                hipEventRecord(event[t+5],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    if(h_FrameParamPtr[0]==0) // Send the updated accumulated frame at the end of the number of counts when it is a preview.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);
    }
    else if(h_FrameParamPtr[0] == 1) // Send the updated accumulated frame in real time when it is acquisition mode.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);    
    }

    hipMemcpy(h_FrameParamPtr,d_FrameParamPtr,11*8,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    /*---- Cleaning ----*/
    
     for(int i=0;i<5;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    
    delete [] stream;
	
    for(int i=0;i<10;i++)
    {
        hipEventDestroy(event[i]);
	}

	delete [] event;
    
    hipFree(d_FrameIndexPtr);
    hipFree(d_BlockCountBuff);
    return cudaStatus;
    
}
