#include "hip/hip_runtime.h"
//////////////////////////////////////////////
//                                          //
//  delocalisation of the data treatement   //
//     on the GPU with CUDA framework       //
//                                          //
//////////////////////////////////////////////

#include "CUDA_Processing.hpp"

/*---------------- GLOBAL VARIABLES TO BE USED ON THE GPU ------------------------------*/




/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------- CUDA KERNELS DEFINITIONS ---------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/


/* struct used to compact the vector */
struct non_negative
{
    __host__ __device__
    bool operator()(const int x)
    {
        return x >= 0;
    }
};



__global__ void InitParam(long *d_FrameParamPtr)
{
    d_FrameParamPtr[6]=0; // number of counts per frame
    __syncthreads();
}

__global__ void ThresholdingData(unsigned char *src,unsigned int *srcAcc, long *d_FrameParamPtr,unsigned int *SSDataStream,unsigned int *SSIndexStream,unsigned int *BlockCountBuff,int t, unsigned char *d_BGCorr)
{
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    int sid=threadIdx.x
    unsigned int BlockThreadoffset;
    unsigned int Blockoffset;
    __shared__ unsigned int BCB;
    __shared__ unisgned int Datash[1000];

    BCB=BlockCountBuff[t];

    long SingleShotRecord=d_FrameParamPtr[7];
    int thresh=(int) d_FrameParamPtr[2];
    long counting_mode=d_FrameParamPtr[4];


    /*--------------------------------------------------------------------------*/
    /*------------------ LOAD ON SHARED MEM & BG SUBSTRACTION ------------------*/
    /*--------------------------------------------------------------------------*/
    
    Datash[sid] =(int) src[id]-(int) d_BGCorr[id];
    __syncthreads();

    /*--------------------------------------------------------------------------*/
    /*--------------------------- THRESHOLDING DATA ----------------------------*/
    /*--------------------------------------------------------------------------*/

    if(Datash[sid]<thresh)
    {
        Datash[sid]=0;
    }
    __syncthreads();


    /*------------------------------------------------------------------------------------*/
    /*------------------ SAVE IN GLOBAL MEM & ACCUMULATE DATA ON FRAME -------------------*/
    /*------------------------------------------------------------------------------------*/

    src[id]=Datash[sid];
    __syncthreads();


    if(SingleShotRecord==0)
    {
        srcAcc[id]+=Datash[sid];
        __syncthreads();

    }

    /*------------------------------------------------------------------------------------*/
    /*-------------------------- END OF ACCUMULATE DATA ON FRAME -------------------------*/
    /*------------------------------------------------------------------------------------*/
    
    
    /*------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
    /*------------------------- VECTOR COMPACTION IN THE KERNEL (ref D.M. Hughes & al. Computer Graphics forum, Vol 32, iss 6, p178-188 (2013)) ------------------------*/
    /*------------------------------------------------------------------------------------------------------------------------------------------------------------------*/
    if(SingleShotRecord==2)
    {
    __shared__ unsigned int W[25];          // array of warp offsets (for 800 threads/block -> 25 warps)
    
    
    unsigned int warpindex=i/32;                    // will take only the floor of the value due to the fact that it is defined as int.
    unsigned int threadindexinwarp= i%32;
    unsigned int threadmask=INT_MAX>>(32-threadindexinwarp); // for example: [1111111111]->[0011111111] if arg=2;
    
    unsigned int ballotresult=__ballot(src[i]) & threadmask;
    unsigned int threadoffset=__popc(ballotresult);
    
    if(threadindexinwarp==31)
    {
        if(src[i]==0)
        {
            W[warpindex]=threadoffset;
        }
        else
        {
            W[warpindex]=threadoffset+1;
        }
        __threadfence();
        __syncthreads();
    }
    
    BlockThreadoffset=__syncthreads_count(src[i]);
    
    unsigned int warpoffset=(__popc(__ballot(W[warpindex] & 1)&threadmask)>>warpindex) +(__popc(__ballot(W[warpindex]&2)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&4)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&8)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&16)&threadmask)>>warpindex)+(__popc(__ballot(W[warpindex]&32)&threadmask)>>warpindex);
    
    Blockoffset=atomicAdd(&BCB,BlockThreadoffset);        // Last problem on that line in compilation.
    BlockCountBuff[t]=BCB;
    unsigned int Totaloffset;
    
    Totaloffset=Blockoffset+warpoffset+threadoffset;
    
    if(src[i]>0)
    {
        SSDataStream[Totaloffset]=src[i];
        SSIndexStream[Totaloffset]=i;
    }
    
    __syncthreads();
    }
    /*-----------------------------------------------------------------------------*/
    /*------------------------- END OF VECTOR COMPACTION --------------------------*/
    /*-----------------------------------------------------------------------------*/
      
}

__global__ void ResetDataArrays(unsigned int *srcAcc)
{
     int i=threadIdx.x+blockIdx.x*blockDim.x;
     srcAcc[i]=0;
     __syncthreads();
}


/*\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-*/


/*----------------------------------------------------------------------------------------*/
/*---------------------------- END OF CUDA KERNELS DEFINITIONS ---------------------------*/
/*----------------------------------------------------------------------------------------*/
/*------------------------------------ DEFINITION OF -------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*-------------------------------- C++ WRAPPING FUNCTIONS --------------------------------*/
/*----------------------------------------------------------------------------------------*/


hipError_t InitialiseCUDAMem(unsigned char **& h_StreamPtr, unsigned int **& d_SSDataStream_ptr, unsigned int **& d_SSIndexStream_ptr, long *& d_FrameParamPtr, unsigned char *& d_FramePtr, unsigned int *& d_PicturePtr, int Nbytes, unsigned char *& d_BGCorr)
{
    hipError_t cudaStatus;
    
    /* Pinned Host memory allocation */

    for(int i=0;i<5;i++)
    {
    cudaStatus=hipHostMalloc((void**)&h_StreamPtr[i],Nbytes);
    }

    //hipHostMalloc((void**)&h_StreamAccPtr,Nbytes*4);
    
    /* Single shot data array allocation */
    /*(for a maximum of 50 millions points which represents 5millions frames at 10 evt/frame (2h45min acq)) */
    
    for(int i=0;i<5;i++)
    {
    hipMalloc((void**) &d_SSDataStream_ptr[i],40000000);
    hipMalloc((void**) &d_SSIndexStream_ptr[i],40000000);
    }
    
    /* memory allocation for frame on GPU */
    
    hipMalloc((void**) &d_FrameParamPtr,11*8);
    hipMalloc((void**) &d_FramePtr,Nbytes);
    hipMalloc((void**) &d_PicturePtr,Nbytes*4);

    hipMalloc((void**) &d_BGCorr,Nbytes);
    
    return cudaStatus;
}

hipError_t FreeCUDAMem(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr, long *d_FrameParamPtr, unsigned char *d_FramePtr, unsigned int *d_PicturePtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    for(int i=0;i<5;i++)
    {
        hipHostFree(h_StreamPtr[i]);
        hipFree(d_SSDataStream_ptr[i]);
        hipFree(d_SSIndexStream_ptr[i]);
    }

    cudaStatus=hipFree(d_FrameParamPtr);
    hipFree(d_FramePtr); // Free the GPU ressources.
    hipFree(d_PicturePtr);
    hipFree(d_BGCorr);
    
    return cudaStatus;
}

hipError_t CUDAresetDataArrays(unsigned int *d_PicturePtr)
{
    hipError_t cudaStatus;
    dim3 grid(400,1);
    dim3 threads(400,1);

    ResetDataArrays<<<grid,threads>>>(d_PicturePtr);
    cudaStatus=hipDeviceSynchronize();
    
    return cudaStatus;
        
}

hipError_t CUDAbackgroundFrameToGPU(unsigned char *h_BGCorr, unsigned char *d_BGCorr, int Nbytes)
{   
    hipError_t cudaStatus;
    cudaStatus=hipMemcpy(d_BGCorr,h_BGCorr,Nbytes,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return cudaStatus;
}

hipError_t CUDAProcessingData(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr,unsigned int *SharedMem, unsigned char *d_FramePtr,  unsigned int *d_PicturePtr, int Nbytes, long *h_FrameParamPtr, long *d_FrameParamPtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    /*---- Cuda streams to optimize data transfer and kernel treatement (5 streams for now) ----*/
    
    hipStream_t *stream=new hipStream_t[5];
    
    for(int i=0;i<5;i++)
    {
        hipStreamCreate(&stream[i]);
    }

    /*---- Buffers for frame Compaction ----*/
    
    unsigned int *d_BlockCountBuff;
    hipMalloc((void**) &d_BlockCountBuff,5*4);
    //__device__ unsigned int d_BlockCountBuff[5];
    
    
    
    /*---- Parameters copy from GUI ----*/
    
    hipMemcpy(d_FrameParamPtr,h_FrameParamPtr,11*8,hipMemcpyHostToDevice);  // copy parameters to treat the data
    
    dim3 grid(160,1);
    dim3 threads(1000,1);
    
    unsigned int* d_FrameIndexPtr;
    hipMalloc((void**) &d_FrameIndexPtr,Nbytes*4);

    hipEvent_t* event=new hipEvent_t[10];

    for(int i=0;i<10;i++)
    {
    hipEventCreate(&event[i]);
	}

    /*---- Core of the processing: data transfer then kernel execution ----*/

    if(h_FrameParamPtr[1]<5)
    {
        for(int t=0;t<h_FrameParamPtr[1];t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);

                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
                    int *vec_compact, *idx_compact;
                    thrust::copy_if(thrust::cuda::par.on(stream[0]), d_FramePtr, d_FramePtr + this->vocab_size , vec_compact, non_negative());
                    //thrust::copy_if(thrust::cuda::par.on(stream[0]), d_FrameIdxPtr, d_FrameIdxPtr + this->vocab_size , idx_compact, non_negative());
                    free(vec_compact);
                    free(idx_compact);

                }

                //hipDeviceSynchronize();
                hipEventRecord(event[h_FrameParamPtr[1]],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
                    int *vec_compact, *idx_compact;
                    thrust::copy_if(thrust::cuda::par.on(stream[t]), d_FramePtr, d_FramePtr + this->vocab_size , vec_compact, non_negative());
                    //thrust::copy_if(thrust::cuda::par.on(stream[t]), d_FrameIdxPtr, d_FrameIdxPtr + this->vocab_size , idx_compact, non_negative());
                    free(vec_compact);
                    free(idx_compact);
                }
                //hipDeviceSynchronize();
                hipEventRecord(event[t+h_FrameParamPtr[1]],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    else
    {
        for(int t=0;t<5;t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);
        
                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks

                // Save data in case of single shot
                if(h_FrameParamPtr[7]==1)
                {
                    int *vec_compact, *idx_compact;
                    thrust::copy_if(thrust::cuda::par.on(stream[0]), d_FramePtr, d_FramePtr + this->vocab_size , vec_compact, non_negative());
                    //thrust::copy_if(thrust::cuda::par.on(stream[0]), d_FrameIdxPtr, d_FrameIdxPtr + this->vocab_size , idx_compact, non_negative());
                    free(vec_compact);
                    free(idx_compact);

                }
                //hipDeviceSynchronize();
                hipEventRecord(event[5],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
                    int *vec_compact, *idx_compact;
                    thrust::copy_if(thrust::cuda::par.on(stream[t]), d_FramePtr, d_FramePtr + this->vocab_size , vec_compact, non_negative());
                    //thrust::copy_if(thrust::cuda::par.on(stream[t]), d_FrameIdxPtr, d_FrameIdxPtr + this->vocab_size , idx_compact, non_negative());
                    free(vec_compact);
                    free(idx_compact);
                }
                //hipDeviceSynchronize();
                hipEventRecord(event[t+5],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    if(h_FrameParamPtr[0]==0) // Send the updated accumulated frame at the end of the number of counts when it is a preview.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);
    }
    else if(h_FrameParamPtr[0] == 1) // Send the updated accumulated frame in real time when it is acquisition mode.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);    
    }

    hipMemcpy(h_FrameParamPtr,d_FrameParamPtr,11*8,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    /*---- Cleaning ----*/
    
     for(int i=0;i<5;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    
    delete [] stream;
	
    for(int i=0;i<10;i++)
    {
        hipEventDestroy(event[i]);
	}

	delete [] event;
    
    hipFree(d_FrameIndexPtr);
    hipFree(d_BlockCountBuff);
    return cudaStatus;
    
}
