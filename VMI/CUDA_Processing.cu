#include "hip/hip_runtime.h"
//////////////////////////////////////////////
//                                          //
//  delocalisation of the data treatement   //
//     on the GPU with CUDA framework       //
//                                          //
//////////////////////////////////////////////

#include "CUDA_Processing.hpp"

/*---------------- GLOBAL VARIABLES TO BE USED ON THE GPU ------------------------------*/




/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------- CUDA KERNELS DEFINITIONS ---------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*----------------------------------------------------------------------------------------*/

__global__ void InitParam(long *d_FrameParamPtr)
{
    d_FrameParamPtr[6]=0; // number of counts per frame
    __syncthreads();
}

__global__ void ThresholdingData(unsigned char *src,unsigned int *srcAcc, long *d_FrameParamPtr,unsigned int *SSDataStream,unsigned int *SSIndexStream,unsigned int *BlockCountBuff,int t, unsigned char *d_BGCorr)
{
    int id=threadIdx.x+blockIdx.x*blockDim.x;
    int sid=threadIdx.x;
    unsigned int BlockThreadoffset;
    unsigned int Blockoffset;
    __shared__ unsigned int BCB;
    __shared__ unsigned int Datash[1000];

    BCB=BlockCountBuff[t];

    long SingleShotRecord=d_FrameParamPtr[7];
    int thresh=(int) d_FrameParamPtr[2];
    long counting_mode=d_FrameParamPtr[4];


    /*--------------------------------------------------------------------------*/
    /*------------------ LOAD ON SHARED MEM & BG SUBSTRACTION ------------------*/
    /*--------------------------------------------------------------------------*/
    
    Datash[sid] =(int) src[id]-(int) d_BGCorr[id];
    __syncthreads();

    /*--------------------------------------------------------------------------*/
    /*--------------------------- THRESHOLDING DATA ----------------------------*/
    /*--------------------------------------------------------------------------*/

    if(Datash[sid]<thresh)
    {
        Datash[sid]=0;
    }
    __syncthreads();


    /*------------------------------------------------------------------------------------*/
    /*------------------ SAVE IN GLOBAL MEM & ACCUMULATE DATA ON FRAME -------------------*/
    /*------------------------------------------------------------------------------------*/

    src[id]=Datash[sid];
    __syncthreads();


    //if(SingleShotRecord==0)
    //{
    srcAcc[id]+=Datash[sid];
    __syncthreads();
    //}

    /*------------------------------------------------------------------------------------*/
    /*-------------------------- END OF ACCUMULATE DATA ON FRAME -------------------------*/
    /*------------------------------------------------------------------------------------*/
    
      
}

__global__ void ResetDataArrays(unsigned int *srcAcc)
{
     int i=threadIdx.x+blockIdx.x*blockDim.x;
     srcAcc[i]=0;
     __syncthreads();
}


/*\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-/-\-*/


/*----------------------------------------------------------------------------------------*/
/*---------------------------- END OF CUDA KERNELS DEFINITIONS ---------------------------*/
/*----------------------------------------------------------------------------------------*/
/*------------------------------------ DEFINITION OF -------------------------------------*/
/*----------------------------------------------------------------------------------------*/
/*-------------------------------- C++ WRAPPING FUNCTIONS --------------------------------*/
/*----------------------------------------------------------------------------------------*/
struct is_not_zero
{
    __host__ __device__
        bool operator()(const int x)
    {
        return (x != 0);
    }
};


hipError_t InitialiseCUDAMem(unsigned char **& h_StreamPtr, unsigned int **& d_SSDataStream_ptr, unsigned int **& d_SSIndexStream_ptr, long *& d_FrameParamPtr, unsigned char *& d_FramePtr, unsigned int *& d_PicturePtr, int Nbytes, unsigned char *& d_BGCorr)
{
    hipError_t cudaStatus;
    
    /* Pinned Host memory allocation */

    for(int i=0;i<5;i++)
    {
    cudaStatus=hipHostMalloc((void**)&h_StreamPtr[i],Nbytes);
    }

    //hipHostMalloc((void**)&h_StreamAccPtr,Nbytes*4);
    
    /* Single shot data array allocation */
    /*(for a maximum of 50 millions points which represents 5millions frames at 10 evt/frame (2h45min acq)) */
    
    for(int i=0;i<5;i++)
    {
    hipMalloc((void**) &d_SSDataStream_ptr[i],40000000);
    hipMalloc((void**) &d_SSIndexStream_ptr[i],40000000);
    }
    
    /* memory allocation for frame on GPU */
    
    hipMalloc((void**) &d_FrameParamPtr,11*8);
    hipMalloc((void**) &d_FramePtr,Nbytes);
    hipMalloc((void**) &d_PicturePtr,Nbytes*4);

    hipMalloc((void**) &d_BGCorr,Nbytes);
    
    return cudaStatus;
}

hipError_t FreeCUDAMem(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr, long *d_FrameParamPtr, unsigned char *d_FramePtr, unsigned int *d_PicturePtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    for(int i=0;i<5;i++)
    {
        hipHostFree(h_StreamPtr[i]);
        hipFree(d_SSDataStream_ptr[i]);
        hipFree(d_SSIndexStream_ptr[i]);
    }

    cudaStatus=hipFree(d_FrameParamPtr);
    hipFree(d_FramePtr); // Free the GPU ressources.
    hipFree(d_PicturePtr);
    hipFree(d_BGCorr);
    
    return cudaStatus;
}

hipError_t CUDAresetDataArrays(unsigned int *d_PicturePtr)
{
    hipError_t cudaStatus;
    dim3 grid(400,1);
    dim3 threads(400,1);

    ResetDataArrays<<<grid,threads>>>(d_PicturePtr);
    cudaStatus=hipDeviceSynchronize();
    
    return cudaStatus;
        
}

hipError_t CUDAbackgroundFrameToGPU(unsigned char *h_BGCorr, unsigned char *d_BGCorr, int Nbytes)
{   
    hipError_t cudaStatus;
    cudaStatus=hipMemcpy(d_BGCorr,h_BGCorr,Nbytes,hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    return cudaStatus;
}

hipError_t CUDAProcessingData(unsigned char **h_StreamPtr, unsigned int **d_SSDataStream_ptr, unsigned int **d_SSIndexStream_ptr,unsigned int *SharedMem, unsigned char *d_FramePtr,  unsigned int *d_PicturePtr, int Nbytes, long *h_FrameParamPtr, long *d_FrameParamPtr, unsigned char *d_BGCorr)
{
    hipError_t cudaStatus;
    
    /*---- Cuda streams to optimize data transfer and kernel treatement (5 streams for now) ----*/
    
    hipStream_t *stream=new hipStream_t[5];
    
    for(int i=0;i<5;i++)
    {
        hipStreamCreate(&stream[i]);
    }

    /*---- Buffers for frame Compaction ----*/
    
    unsigned int *d_BlockCountBuff;
    hipMalloc((void**) &d_BlockCountBuff,5*4);
    //__device__ unsigned int d_BlockCountBuff[5];
    
    
    
    /*---- Parameters copy from GUI ----*/
    
    hipMemcpy(d_FrameParamPtr,h_FrameParamPtr,11*8,hipMemcpyHostToDevice);  // copy parameters to treat the data
    
    dim3 grid(160,1);
    dim3 threads(1000,1);
    
    unsigned int* d_FrameIndexPtr;
    hipMalloc((void**) &d_FrameIndexPtr,Nbytes*4);

    hipEvent_t* event=new hipEvent_t[10];

    for(int i=0;i<10;i++)
    {
    hipEventCreate(&event[i]);
	}

    /*---- Core of the processing: data transfer then kernel execution ----*/

    if(h_FrameParamPtr[1]<5)
    {
        for(int t=0;t<h_FrameParamPtr[1];t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);

                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
                    thrust::device_ptr<unsigned char> d_FramePtrVec = thrust::device_pointer_cast(d_FramePtr);
					thrust::device_vector<unsigned char> d_FrameVecCompact(Nbytes);
                    auto result_end=thrust::copy_if(thrust::cuda::par.on(stream[t]),d_FramePtrVec, d_FramePtrVec+400*400,d_FrameVecCompact.begin(),is_not_zero());
					//thrust::host_vector<unsigned char> h_FrameVecCompact(d_FrameVecCompact.begin(),result_end);
					//thrust::copy(h_FrameVecCompact.begin(),h_FrameVecCompact.end(),std::ostream_iterator<unsigned char>(SSofile, "\n"));
					//thrust::copy_if(d_FramePtrVec-, d_FramePtrVec + 400*400 , std::ostream_iterator<unsigned char>(SSofile, "\n"));
                    //thrust::copy_if(thrust::cuda::par, d_FrameIdxPtrVec, d_FrameIdxPtrVec + 400*400 , std::ostream_iterator<int>(SSofile, "\n"), is_not_zero());
					
                }

                //hipDeviceSynchronize();
                hipEventRecord(event[h_FrameParamPtr[1]],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
                    thrust::device_ptr<unsigned char> d_FramePtrVec = thrust::device_pointer_cast(d_FramePtr);
					thrust::device_vector<unsigned char> d_FrameVecCompact(Nbytes);
                    auto result_end=thrust::copy_if(thrust::cuda::par.on(stream[t]),d_FramePtrVec, d_FramePtrVec+400*400,d_FrameVecCompact.begin(),is_not_zero());
					//thrust::host_vector<unsigned char> h_FrameVecCompact(d_FrameVecCompact.begin(),result_end);
					
                }
                //hipDeviceSynchronize();
                hipEventRecord(event[t+h_FrameParamPtr[1]],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    else
    {
        for(int t=0;t<5;t++) // Create Asynchronous data transfer and kernel
        {
            if(t==0)
            {
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[0],stream[0]);
        
                InitParam<<<1,1,0,stream[0]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks

                // Save data in case of single shot
                if(h_FrameParamPtr[7]==1)
                {
					thrust::device_ptr<unsigned char> d_FramePtrVec = thrust::device_pointer_cast(d_FramePtr);
					thrust::device_vector<unsigned char> d_FrameVecCompact(Nbytes);
                    auto result_end=thrust::copy_if(thrust::cuda::par.on(stream[t]),d_FramePtrVec, d_FramePtrVec+400*400,d_FrameVecCompact.begin(),is_not_zero());
					//thrust::host_vector<unsigned char> h_FrameVecCompact(d_FrameVecCompact.begin(),result_end);

                }
                //hipDeviceSynchronize();
                hipEventRecord(event[5],stream[0]);
            }
            else
            {
                hipStreamWaitEvent(stream[t],event[t-1],0);
                hipMemcpyAsync(d_FramePtr,h_StreamPtr[t],Nbytes,hipMemcpyHostToDevice,stream[t]);       // Copy the frame to the GPU
                //hipDeviceSynchronize();
                hipEventRecord(event[t],stream[t]);

                hipStreamWaitEvent(stream[t],event[t+1],0);
                InitParam<<<1,1,0,stream[t]>>>(d_FrameParamPtr);
                ThresholdingData<<<grid,threads,0,stream[t]>>>(d_FramePtr,d_PicturePtr,d_FrameParamPtr,d_SSDataStream_ptr[t],d_SSIndexStream_ptr[t],d_BlockCountBuff,t,d_BGCorr);     // Call thresholding data with number of blocks
                if(h_FrameParamPtr[7]==1)
                {
					thrust::device_ptr<unsigned char> d_FramePtrVec = thrust::device_pointer_cast(d_FramePtr);
					thrust::device_vector<unsigned char> d_FrameVecCompact(Nbytes);
                    auto result_end=thrust::copy_if(thrust::cuda::par.on(stream[t]),d_FramePtrVec, d_FramePtrVec+400*400,d_FrameVecCompact.begin(),is_not_zero());
					//thrust::host_vector<unsigned char> h_FrameVecCompact(d_FrameVecCompact.begin(),result_end);
                }
                //hipDeviceSynchronize();
                hipEventRecord(event[t+5],stream[t]);
            }
            hipStreamSynchronize(stream[t]);
        }
        //hipDeviceSynchronize();
    }
    if(h_FrameParamPtr[0]==0) // Send the updated accumulated frame at the end of the number of counts when it is a preview.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);
    }
    else if(h_FrameParamPtr[0] == 1) // Send the updated accumulated frame in real time when it is acquisition mode.
    {
        cudaStatus=hipMemcpy(SharedMem,d_PicturePtr,Nbytes*4,hipMemcpyDeviceToHost);    
    }

    hipMemcpy(h_FrameParamPtr,d_FrameParamPtr,11*8,hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    /*---- Cleaning ----*/
    
     for(int i=0;i<5;i++)
    {
        hipStreamDestroy(stream[i]);
    }
    
    delete [] stream;
	
    for(int i=0;i<10;i++)
    {
        hipEventDestroy(event[i]);
	}

	delete [] event;
    
    hipFree(d_FrameIndexPtr);
    hipFree(d_BlockCountBuff);
    return cudaStatus;
    
}
